#include "hip/hip_runtime.h"
#include "../source/Paging.cuh"
#include <iostream>

__global__ void writeKernel(int pageID){
	// write 4-byte integers from 1 to PAGE_SIZE/4 to each page
	int pageSize = PAGE_SIZE_DEFAULT;
	int *arr = (int*)pageAddress(pageID);
	for (int i=0; i<pageSize/4; i++){
		arr[i] = i+1;
	}
}

__global__ void readKernel(int pageID){
	// check if the integers on the page are consistent with the write
	int *arr = (int*)pageAddress(pageID);
	int pageSize = PAGE_SIZE_DEFAULT;
	for (int i=0; i<pageSize/4; i++){
		if (arr[i] != (i+1))
			__trap();
	}
}

int main(int argc, char const *argv[])
{
	/* init Buffer */
	initPages();
	/* test reading and writing on all pages */
	fprintf(stderr, "testing ... \n");
	for (unsigned pageID=0; pageID<(TOTAL_N_PAGES_DEFAULT); pageID++){
		writeKernel <<< 1, 1 >>> (pageID);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		readKernel <<< 1, 1 >>> (pageID);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		// display progress
		float progress = (float)pageID/(TOTAL_N_PAGES_DEFAULT);
		std::cerr << " %\r" << int(progress * 100.0);
		// std::cerr << "[";
		// int barWidth = 70;
		// int pos = barWidth * progress;
		// for (int i = 0; i < barWidth; ++i) {
		// 	if (i < pos) std::cerr << "=";
		// 	else if (i == pos) std::cerr << ">";
		// 	else std::cerr << " ";
		// }
		// std::cerr << "] " << int(progress * 100.0) << " %\r";
		std::cerr.flush();
	}

	std::cerr << "\n done \n" ;

	return 0;
}