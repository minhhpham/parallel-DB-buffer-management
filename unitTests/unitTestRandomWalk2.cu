#include "hip/hip_runtime.h"
#include "parallelPage.cuh"
#include "metrics.h"

static __device__ void fillPage(void *page){
	char *ptr = (char*)page;
	for (int i=0; i<PAGE_SIZE; i++)
		ptr[i] = 1;
}

/* Kernel to get 1 page with Random Walk, record step counts */
__global__ void RandomWalk_get1page_kernel(int Nthreads, int *d_step_counts){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid<Nthreads){
		int step_counts;
		int *tmp = d_step_counts? &step_counts : 0;
		int pageID = getPageRandomWalk(tmp);
		if (d_step_counts) d_step_counts[tid] = step_counts;
		// mem check
		// void *page = pageAddress(pageID);
		// fillPage(page);
	}
}


/* Execute one kernel of N threads, each gets 1 page with Random Walk
	input: Nthreads: 		number of threads
	return: *avgStep:		average of step counts across all threads
			*avgMaxWarp:	average of Max of Warp across all warps
			*runTime:		total run time (s)
 */
Metrics_t runRandomWalk(int Nthreads, int NFree){
	// allocate metrics array on host
	int *h_step_counts = (int*)malloc(10000*sizeof(int));
	// allocate metrics array on gpu
	int *d_step_counts;
	gpuErrchk( hipMalloc((void**)&d_step_counts, 10000*sizeof(int)) );

	// run kernel until get to NFree
	resetBufferRandomWalk();
	// printNumPagesLeftRandomWalk();
	int NGets = TOTAL_N_PAGES - NFree;
	for (int i=0; i<(NGets/5000); i++){
		RandomWalk_get1page_kernel <<< ceil((float)5000/32), 32 >>> (5000, 0);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
	}

	for (int i=0; i<(NGets-(NGets/5000)*5000)/1000; i++){
		RandomWalk_get1page_kernel <<< ceil((float)1000/32), 32 >>> (1000, 0);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
	}

	// printNumPagesLeftRandomWalk();
	// execute kernel;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	RandomWalk_get1page_kernel <<< ceil((float)Nthreads/32), 32 >>> (Nthreads, d_step_counts);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float total_time;
	hipEventElapsedTime(&total_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// copy metrics to host
	gpuErrchk( hipMemcpy(h_step_counts, d_step_counts, Nthreads*sizeof(int), hipMemcpyDeviceToHost) );

	// aggregate metrics and return
	Metrics_t out = aggregate_metrics(h_step_counts, Nthreads);
	out.runTime = total_time;
	free(h_step_counts); hipFree(d_step_counts);
	return out;
}


int main(int argc, char const *argv[])
{
	gpuErrchk(hipSetDevice(0));
	/* command descriptions */
	if(argc>1 && ((strncmp(argv[1], "-h", 2) == 0) || (strncmp(argv[1], "-help", 4) == 0))){
		fprintf(stderr, "USAGE: ./unitTestRandomWalk [options]\n");
		fprintf(stderr, "OPTIONS:\n");
		// fprintf(stderr, "\t -pn, --pageNum <pageNum>\n");
		// fprintf(stderr, "\t\t Total Pages, default is 1000000\n\n");

		fprintf(stderr, "\t -tn, --threadNum <threadsNum>\n");
		fprintf(stderr, "\t\t Total threads that are asking pages, default is 5000\n\n");

		// fprintf(stderr, "\t -lp, --leftPage <leftPageNum>\n");
		// fprintf(stderr, "\t\t Pages left in the system, default is all pages free \n\n");
	}

	/* parse options */
	int Nthreads=0;
	for (int i=0; i<argc; i++){
		if((strncmp(argv[i], "-tn", 3) == 0) || (strcmp(argv[i], "--threadNum") == 0))
			Nthreads = atoi(argv[i]);

	}
	if (Nthreads==0) Nthreads = 5000;


	/* initialize system, all pages free, parameters defined in parallelPage.cuh */
	fprintf(stderr, "initializing page system ... \n");
	initPagesRandomWalk();
	printNumPagesLeftRandomWalk();

	Metrics_t *metrics_array = (Metrics_t*)malloc(100*sizeof(Metrics_t));

	/* repeat getpage with Random Walk */
	fprintf(stderr, "unit test with Total Pages = %d, Nthreads = %d ...\n", TOTAL_N_PAGES, Nthreads);
	Nthreads = 5000;
	printf("T,N,A,Average_steps,Average_Max_Warp,Time(ms)\n");
	for (int AvailablePages=700000; AvailablePages>=500000; AvailablePages-=5000){
		printf("%d,%d,%d,", TOTAL_N_PAGES, Nthreads, AvailablePages);

		// run 100 times
		Metrics_t metrics_array[100];
		for (int trial=0; trial<100; trial++){
			metrics_array[trial] = runRandomWalk(Nthreads, AvailablePages);
			// printf("%f\n", metrics_array[trial].runTime);
		}

		// take avg
		float avgStep = 0;
		float avgMaxWarp = 0;
		float avgTime = 0;
		for (int i=0; i<100; i++){
			avgStep+=metrics_array[i].avgStep;
			avgMaxWarp+=metrics_array[i].avgMaxWarp;
			avgTime+=metrics_array[i].runTime;
		}
		avgStep = avgStep/100;
		avgMaxWarp = avgMaxWarp/100;
		avgTime = avgTime/100;


		// print results to stdout
		printf("%f,%f,%f\n", avgStep, avgMaxWarp, avgTime);
	}	

	return 0;
}