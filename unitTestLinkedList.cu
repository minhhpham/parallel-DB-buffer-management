#include "hip/hip_runtime.h"
#include "parallelPage.cuh"
#include "metrics.h"

static __device__ void fillPage(void *page){
	char *ptr = (char*)page;
	for (int i=0; i<PAGE_SIZE; i++)
		ptr[i] = 1;
}

/* Kernel to get 1 page with Random Walk, record step counts */
__global__ void LinkedList_get1page_kernel(int Nthreads, int *d_step_counts){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid<Nthreads){
		int step_counts;
		int pageID = getPageLinkedList(&step_counts);
		// freePageLinkedList(pageID);
		// pageID = getPageLinkedList(&step_counts);
		// d_step_counts[tid] = step_counts;
		// mem check
		void *page = pageAddress(pageID);
		fillPage(page);
	}
}


/* Execute one kernel of N threads, each gets 1 page with Random Walk
	input: Nthreads: 		number of threads
	return: *avgStep:		average of step counts across all threads
			*avgMaxWarp:	average of Max of Warp across all warps
			*runTime:		total run time (s)
 */
Metrics_t runLinkedList(int Nthreads){
	// allocate metrics array on host
	int *h_step_counts = (int*)malloc(Nthreads*sizeof(int));
	// allocate metrics array on gpu
	int *d_step_counts;
	gpuErrchk( hipMalloc((void**)&d_step_counts, Nthreads*sizeof(int)) );

	// execute kernel;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	LinkedList_get1page_kernel <<< ceil((float)Nthreads/32), 32 >>> (Nthreads, d_step_counts);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float total_time;
	hipEventElapsedTime(&total_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printNumPagesLeftLinkedList();

	// copy metrics to host
	gpuErrchk( hipMemcpy(h_step_counts, d_step_counts, Nthreads*sizeof(int), hipMemcpyDeviceToHost) );

	// aggregate metrics and return
	Metrics_t out = aggregate_metrics(h_step_counts, Nthreads);
	out.runTime = total_time;
	free(h_step_counts); hipFree(d_step_counts);
	return out;
}


int main(int argc, char const *argv[])
{
	gpuErrchk(hipSetDevice(0));
	/* command descriptions */
	if(argc>1 && ((strncmp(argv[1], "-h", 2) == 0) || (strncmp(argv[1], "-help", 4) == 0))){
		fprintf(stderr, "USAGE: ./unitTestRandomWalk [options]\n");
		fprintf(stderr, "OPTIONS:\n");
		// fprintf(stderr, "\t -pn, --pageNum <pageNum>\n");
		// fprintf(stderr, "\t\t Total Pages, default is 1000000\n\n");

		fprintf(stderr, "\t -tn, --threadNum <threadsNum>\n");
		fprintf(stderr, "\t\t Total threads that are asking pages, default is 5000\n\n");

		// fprintf(stderr, "\t -lp, --leftPage <leftPageNum>\n");
		// fprintf(stderr, "\t\t Pages left in the system, default is all pages free \n\n");
	}

	/* parse options */
	int Nthreads=0;
	for (int i=0; i<argc; i++){
		if((strncmp(argv[i], "-tn", 3) == 0) || (strcmp(argv[i], "--threadNum") == 0))
			Nthreads = atoi(argv[i]);

	}
	if (Nthreads==0) Nthreads = 5000;


	/* initialize system, all pages free, parameters defined in parallelPage.cuh */
	fprintf(stderr, "initializing page system ... \n");
	initPagesLinkedList();
	printNumPagesLeftLinkedList();

	/* repeat getpage with Random Walk */
	fprintf(stderr, "unit test with Total Pages = %d, Nthreads = %d ...\n", TOTAL_N_PAGES, Nthreads);
	int AvailablePages = TOTAL_N_PAGES;
	printf("T,N,A,Average_steps,Average_Max_Warp,Time(ms)\n");
	for (int i=0; i<TOTAL_N_PAGES/Nthreads; i++){
		// run kernel to get 1 page for each thread
		Metrics_t metrics = runLinkedList(Nthreads);
		// print results to stdout
		printf("%d,%d,%d,%f,%f,%f\n", TOTAL_N_PAGES, Nthreads, AvailablePages, metrics.avgStep, metrics.avgMaxWarp, metrics.runTime);
		AvailablePages-=Nthreads;
	}

	return 0;
}