#include "hip/hip_runtime.h"
#include "parallelPage.cuh"
#include <stdint.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>

namespace cg = cooperative_groups;

#define GROUP_N_PAGES 100000   // group pages for faster initialization

/* actual pages for all strategies */
__device__ void **d_page_groups;                                       // actual address to pages, (TOTAL_N_PAGES/GROUP_N_PAGES) groups, each with GROUP_N_PAGES pages
__device__ void *pageAddress(int pageID){
	// find group id
	int groupID = pageID/GROUP_N_PAGES;
	// find offset on group
	int offset = pageID - groupID*GROUP_N_PAGES;
	// calculate actual address
	void *groupAddr = d_page_groups[groupID];
	return (void*)((char*)groupAddr + offset*PAGE_SIZE);
}


/* data structures for Random Walk*/
// for each element: 0 means page is free, 1 means page is occupied
static __device__ int *d_PageMapRandomWalk;
static __device__ int *d_RNG;
static __device__ int *d_RNG_idx;

/* data structures for Clustered Random Walk */
static __device__ int *d_LastFreePage;	// last free pageID obtained by each thread by calling freePageClusteredRandomWalk(), -1 if not initialized

/* data structures for Linked List */
typedef struct Node_t
{
	int pageID;
	Node_t *nextNode;	// null if none
} Node_t;
static __device__ Node_t *d_nodes;		// pre-allocated nodes
volatile static __device__ int d_HeadNodeID;		// index of headnode on d_nodes, -1 if it's not yet returned by a thread, -2 if no free page
volatile static __device__ int d_TailNodeID;		// index of tailnode on d_nodes, -1 if it's not yet returned by a thread

static __device__ long long unsigned d_LLticket;
static __device__ long long unsigned d_LLturn;
// static __device__ Node_t *d_HeadNode;	// pointer to first free node
// static __device__ int d_lockHeadNode;	// 0 means free
// static __device__ Node_t *d_TailNode;	// pointer to last free node
// static __device__ int d_lockTailNode;	// 0 means free

/* actual page initialization on GPU */
__host__ void initPages(){
	int n_groups = ceil((float)TOTAL_N_PAGES/GROUP_N_PAGES);
	printf("initializing %d groups on GPU, each with %d pages. Total = %ld MB \n", n_groups, GROUP_N_PAGES, (long)n_groups*GROUP_N_PAGES*PAGE_SIZE/1048576);
	// initialize (TOTAL_N_PAGES/GROUP_N_PAGES) groups on GPU memory, save their pointers on a host array
	void **h_groups = (void**)malloc(n_groups*sizeof(void*));
	for (int i=0; i<n_groups; i++){ // allocate each group
		gpuErrchk( hipMalloc((void**)&h_groups[i], GROUP_N_PAGES*PAGE_SIZE) );
	}
	// allocate an array for d_pages and transfer the array of group pointers to this array
	void* tmp;
	hipMalloc(&tmp, n_groups*sizeof(void*));
	gpuErrchk( hipMemcpy(tmp, h_groups, n_groups*sizeof(void*), hipMemcpyHostToDevice) );

	// set d_pages to this array
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_page_groups), &tmp, sizeof(void*)) );

	// end
	free(h_groups);
}


/* RANDOM WALK IMPLEMENTATION */

/* Arrange the N elements of ARRAY in random order.
   Only effective if N is much smaller than RAND_MAX;
   if this may not be the case, use a better random
   number generator. */
static inline void shuffle(int *array, size_t n)
{
    if (n > 1) 
    {
        size_t i;
        for (i = 0; i < n - 1; i++) 
        {
          size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
          int t = array[j];
          array[j] = array[i];
          array[i] = t;
        }
    }
}

/* function to replace hiprand, better efficiency */
#define LCG_M 1<<31
#define LCG_A 1103515245
#define LCG_C 12345
__device__ static inline int RNG_LCG(int seed){
    long long seed_ = (long long)seed;
    return (int)((LCG_A*seed_ + LCG_C)%(LCG_M));
    // uint32_t x = seed;
    // x ^= x << 13;
    // x ^= x >> 17;
    // x ^= x << 5;
    // return x;
}

/* initialize TOTAL_N_PAGES pages on GPU's global memory, each page is PAGE_SIZE large
also initialize the page map structure with all 0 (all free) */
__host__ void initPagesRandomWalk(){
	// initialize actual pages
	initPages();

	// initialize page map
	void *h_PageMapRandomWalk;
	gpuErrchk( hipMalloc((void**)&h_PageMapRandomWalk, TOTAL_N_PAGES*sizeof(int)) );
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_PageMapRandomWalk), &h_PageMapRandomWalk, sizeof(void*)) );
	gpuErrchk( hipMemset(h_PageMapRandomWalk, 0, TOTAL_N_PAGES*sizeof(int)) );

	// initialize random numbers
	int *h_RNG;
	gpuErrchk( hipMalloc((void**)&h_RNG, TOTAL_N_PAGES*sizeof(int)) ); // allocate memory on device then copy to symbol
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_RNG), &h_RNG, sizeof(void*)) );
	srand(time(NULL));
	int *tmp = (int*)malloc(TOTAL_N_PAGES*sizeof(int));
	for (int i=0; i<TOTAL_N_PAGES; i++)
		tmp[i] = i;
	shuffle(tmp, TOTAL_N_PAGES);
	// copy rng numbers to device
	gpuErrchk( hipMemcpy(h_RNG, tmp, TOTAL_N_PAGES*sizeof(int), hipMemcpyHostToDevice) );

	// initialize RNG index
	for (int i=0; i<TOTAL_N_PAGES; i++)
		tmp[i] = 0;
	// copy RNG index to device
	int *h_RNG_idx;
	gpuErrchk( hipMalloc((void**)&h_RNG_idx, TOTAL_N_PAGES*sizeof(int)) ); // allocate memory on device then copy to symbol
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_RNG_idx), &h_RNG_idx, sizeof(void*)) );
	gpuErrchk( hipMemcpy(h_RNG_idx, tmp, TOTAL_N_PAGES*sizeof(int), hipMemcpyHostToDevice) );

	free(tmp);
}



// if step_count is not null, write step count to it
__device__ int d_counter = 0;
__device__ int getPageRandomWalk(int *stepCount){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int16_t Clock = (int16_t)clock();
	int seed = (tid<<15) + Clock;
	seed = RNG_LCG(seed);
	// randomize pages and try
	int pageID = seed % TOTAL_N_PAGES;
	if (pageID<0) pageID = -pageID;
	int step_count = 1;
	while(atomicExch(&d_PageMapRandomWalk[pageID],1) == 1){
		seed = RNG_LCG(seed);
		pageID = seed % TOTAL_N_PAGES;
		step_count++;
	}

	// int pageID = tid*100;
	// atomicExch(&d_PageMapRandomWalk[pageID],1);
	// int step_count = 1;

	if (stepCount) *stepCount = step_count;
	return pageID;
}


__device__ void freePageRandomWalk(int pageID){
	atomicExch(&d_PageMapRandomWalk[pageID], 0);
}


__global__ void resetBufferRandomWalk_kernel(){
	memset(d_PageMapRandomWalk, 0, TOTAL_N_PAGES*sizeof(int));
}

__host__ void resetBufferRandomWalk(){
	resetBufferRandomWalk_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

__global__ void printNumPagesLeftRandomWalk_kernel(){
	int count = 0;
	for (int i=0; i<TOTAL_N_PAGES; i++){
		if (d_PageMapRandomWalk[i]==0) count++;
	}
	printf("[RW info] Number of free pages: %d \n", count);
}

__host__ void printNumPagesLeftRandomWalk(){
	printNumPagesLeftRandomWalk_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

/* CLUSTERED RANDOM WALK IMPLEMENTATION */
/*
 - initialize TOTAL_N_PAGES pages on GPU's global memory, each page is PAGE_SIZE large
 - initialize the page map structure with all 0 (all free) 
 - initialize the d_LastFreePage array to -1
 */
__host__ void initPagesClusteredRandomWalk(){
	initPagesRandomWalk();

	// init the last free page array
	void *d_tmp;	// global array, set to all -1
	gpuErrchk( hipMalloc((void**)&d_tmp, 65536*1024*sizeof(int)) );	// allocate up to a grid's max number of threads
	gpuErrchk( hipMemset(d_tmp, -1, TOTAL_N_PAGES*sizeof(int)) );
	// copy ptr value to symbol d_LastFreePage
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_LastFreePage), &d_tmp, sizeof(void*)) );
}

// if step_count is not null, write step count to it
__device__ int getPageClusteredRandomWalk(int *stepCount){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	// check page next to the last page visited
	int pageID = d_LastFreePage[tid];
	if (pageID!=-1){
		pageID++;
		if (pageID==TOTAL_N_PAGES) pageID = 0;
		if (d_PageMapRandomWalk[pageID]==0){
			if (atomicExch(&d_PageMapRandomWalk[pageID],1) == 0){	// success
				if (stepCount) *stepCount = 1;
				d_LastFreePage[tid] = pageID;
				return pageID;
			}
		}
	}
	// regular random walk
	pageID = getPageRandomWalk(stepCount);
	d_LastFreePage[tid] = pageID;
	return pageID;
}

// same free page function
__device__ void freePageClusteredRandomWalk(int pageID){
	atomicExch(&d_PageMapRandomWalk[pageID], 0);
}

__global__ void printNumPagesLeftClusteredRandomWalk_kernel(){
	// count free pages
	int count = 0;
	for (int i=0; i<TOTAL_N_PAGES; i++){
		if (d_PageMapRandomWalk[i]==0) count++;
	}
	printf("[CRW info] Number of free pages: %d \n", count);
	// count clusters
	int state = d_PageMapRandomWalk[0];
	if (state==0) count = 1;
	else count = 0;
	for (int i=1; i<TOTAL_N_PAGES; i++){
		if (state==0 && d_PageMapRandomWalk[i]==1)	// end of cluster
			state = 1;
		if (state==1 && d_PageMapRandomWalk[i]==0){	// start of cluster
			state = 0;
			count++;
		}
	}
	// check last page and first page if they are in one cluster
	if (state==0 && d_PageMapRandomWalk[0]==0) count--;
	printf("[CRW info] Number of clusters: %d \n", count);
}

__host__ void printNumPagesLeftClusteredRandomWalk(){
	printNumPagesLeftClusteredRandomWalk_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}


/* LINKED LIST IMPLEMENTATION */
/* initialize TOTAL_N_PAGES pages on GPU's global memory, each page is PAGE_SIZE large
	also initialize the linked list structure, all pages free */
__global__ void initPagesLinkedList_kernel(){
	int i = blockIdx.x*blockDim.x + threadIdx.x; // nodeID
	if (i>=TOTAL_N_PAGES) return;
	d_nodes[i].pageID = i;	// pageID = nodeID
	if (i<TOTAL_N_PAGES-1)
		d_nodes[i].nextNode = &d_nodes[i+1];
	else
		d_nodes[i].nextNode = NULL; // last node has no next
	// set head node
	if (i==0)
		d_HeadNodeID = 0;	// first node is head
	// set tail node
	if (i==TOTAL_N_PAGES-1)	// last node is tail
		d_TailNodeID = TOTAL_N_PAGES-1;

	// init lock
	if (i==0){
		d_LLticket = 0;
		d_LLturn = 0;
	}
}

__host__ void initPagesLinkedList(){
	// initialize pages
	initPages();

	// initialize linked list 
	Node_t *h_d_nodes;	// allocate nodes array and get pointer value on host, then copy this value to d_nodes
	gpuErrchk( hipMalloc((void**)&h_d_nodes, TOTAL_N_PAGES*sizeof(Node_t)) );
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_nodes), &h_d_nodes, sizeof(Node_t*)) );

	initPagesLinkedList_kernel <<< ceil((float)TOTAL_N_PAGES/32), 32 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

// if step_count is not null, write step count to it
// atomically get d_HeadNodeID and replace with -1 until one thread replace it with a legit nodeID
// return -2 to d_HeadNodeID if no page is free
__device__ int getPageLinkedList(int *stepCount){
	unsigned long long start = (unsigned long long)clock();
	int nodeID = -1;
	int pageID;
	int step_count = 0;

	while (nodeID==-1){
		nodeID = atomicExch((int*)&d_HeadNodeID, -1);
		if (nodeID != -1){	// this thread successfully capture the head node
			// check if OOM
			if (nodeID==-2){printf("Out of Free Page!\n"); __trap();}
			// find pageID, should be equal to nodeID
			pageID = d_nodes[nodeID].pageID;
			if (pageID!=nodeID){printf("Fatal Error in getPageLinkedList: Inconsistent Node Data\n"); __trap();}
			int next_nodeID;
			if (d_nodes[nodeID].nextNode==NULL) next_nodeID = -2;	// no free page
			else next_nodeID = d_nodes[nodeID].nextNode->pageID;
			// invalidate obtained node's nextNode
			d_nodes[nodeID].nextNode = NULL;
			// return the next legit nodeID back to d_HeadNodeID
			atomicExch((int*)&d_HeadNodeID, next_nodeID);
		}
		
		step_count++;
	}
	unsigned long long stop = (unsigned long long)clock();
	if (stepCount) *stepCount = (int)(stop - start);
	return pageID;

	// FA MUTEX
	// volatile unsigned long long start = (unsigned long long)clock();
	// int pageID;
	// cg::grid_group grid = cg::this_grid();

	// volatile unsigned long long thread_ticket;
	// thread_ticket = atomicAdd(&d_LLticket, (unsigned long long)1);
	// grid.sync();

	// // access linked list
	// do {
	// 	if (thread_ticket == d_LLturn){
	// 		// find pageID, should be equal to nodeID
	// 		pageID = d_HeadNodeID;
	// 		d_HeadNodeID = d_nodes[pageID].nextNode->pageID;
	// 		// increase turn
	// 		atomicAdd(&d_LLturn, (unsigned long long)1);
	// 	}
	// 	grid.sync();
	// } while (d_LLturn<d_LLticket);
	// unsigned long long stop = (unsigned long long)clock();
	// if (stepCount) *stepCount = (int)(stop - start);
	// return 0;
}


// atomically get d_TailNodeID and replace with -1 until one thread replace it with a legit nodeID
// if successfully obtained the tail node (!=-1), replace it with the new tail(=pageID)
__device__ void freePageLinkedList(int pageID, int *stepCount){
	unsigned long long start = (unsigned long long)clock();
	volatile int nodeID = -1;
	volatile int pageID_ = pageID;

	while (nodeID==-1){
		nodeID = atomicExch((int*)&d_TailNodeID, -1);
		if (nodeID != -1){	// this thread successfully capture the tail node
			volatile Node_t *current_node = &d_nodes[pageID_];
			current_node->nextNode = NULL;	// probably redundant
			// now nodeID is the index to tail node
			volatile Node_t *tail_node = &d_nodes[nodeID];
			// update tailnode's next
			tail_node->nextNode = (Node_t*)current_node;
			// return new tail node (pageID) to d_TailNodeID
			atomicExch((int*)&d_TailNodeID, (int)pageID_);
		}
	}
	unsigned long long stop = (unsigned long long)clock();
	if (stepCount) *stepCount = (int)(stop - start);

	// FA MUTEX
	// cg::grid_group grid = cg::this_grid();
	// volatile unsigned long long thread_ticket;
	// thread_ticket = atomicAdd(&d_LLticket, (unsigned long long)1);
	// grid.sync();
	// // access linked list
	// do {
	// 	if (thread_ticket == d_LLturn){
	// 		Node_t *current_node = &d_nodes[pageID];
	// 		current_node->nextNode = NULL;
	// 		// update current tail's node
	// 		Node_t *tail_node = &d_nodes[d_TailNodeID];
	// 		tail_node->nextNode = (Node_t*)current_node;
	// 		// make current node tail
	// 		d_TailNodeID = pageID;
	// 		// increase turn
	// 		atomicAdd(&d_LLturn, (unsigned long long)1);
	// 	}
	// 	grid.sync();
	// } while (d_LLturn<d_LLticket);
}

void resetBufferLinkedList(){
	initPagesLinkedList_kernel <<< ceil((float)TOTAL_N_PAGES/32), 32 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

__global__ void printNumPagesLeftLinkedList_kernel(){
	if (d_HeadNodeID<0){
		printf("[LL info] no free page\n");
		return;
	}
	Node_t *node = &d_nodes[d_HeadNodeID];
	int count = 0;
	while (node){
		count++;
		node = node->nextNode;
	}
	printf("[LL info] Number of pages in linked list: %d, start=%d, end=%d \n", count, d_HeadNodeID, d_TailNodeID);
}

__host__ void printNumPagesLeftLinkedList(){
	printNumPagesLeftLinkedList_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}



/* ------------------------------------- Single Clock Implementation -------------------------------------------- */
static __device__ int d_SingleClockArm;		// index to page number, from 0 to TOTAL_N_PAGES-1
static __device__ int *d_pageMapSingleClock;

/* initialize TOTAL_N_PAGES pages on GPU's global memory, each page is PAGE_SIZE large
also initialize the page map structure with all 0 (all free)
set clock arm to 0
 */
__host__ void initPagesSingleClock(){
	// initialize actual pages
	initPages();
	// initialize page map
	void *h_pageMapSingleClock;
	gpuErrchk( hipMalloc((void**)&h_pageMapSingleClock, TOTAL_N_PAGES*sizeof(int)) );
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_pageMapSingleClock), &h_pageMapSingleClock, sizeof(void*)) );
	gpuErrchk( hipMemset(h_pageMapSingleClock, 0, TOTAL_N_PAGES*sizeof(int)) );
	// set clock arm to 0
	int zero = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_SingleClockArm), &zero, sizeof(int)) );
}

__device__ int getPageSingleClock(int *step_count){
	bool foundFreePage = false;
	int pageID;
	int stepCount = 0;
	while (!foundFreePage){
		stepCount++;
		// atomically shift the arm
		pageID = atomicAdd(&d_SingleClockArm, 1);
		pageID = pageID % TOTAL_N_PAGES;
		// check if the obtained position is available
		if (d_pageMapSingleClock[pageID] == 0){
			foundFreePage = true;
			// mark page as used
			atomicExch(&d_pageMapSingleClock[pageID], 1);
		}
	}
	if (step_count) *step_count = stepCount;
	return pageID;
}

__device__ void freePageSingleClock(int pageID, int *step_count){
	atomicExch(&(d_pageMapSingleClock[pageID]), 0);
}

__global__ void printNumPagesLeftSingleClock_kernel(){
	int count = 0;
	for (int i=0; i<TOTAL_N_PAGES; i++){
		if (d_pageMapSingleClock[i]==0) count++;
	}
	printf("[SC info] Number of free pages: %d \n", count);
}
__host__ void printNumPagesLeftSingleClock(){
	printNumPagesLeftSingleClock_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

__global__ void resetBufferSingleClock_kernel(){
	memset(d_pageMapSingleClock, 0, TOTAL_N_PAGES*sizeof(int));
}
__host__ void resetBufferSingleClock(){
	resetBufferSingleClock_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

/* ------------------------------------- Parallel Clock Implementation -------------------------------------------- */
#define MAX_THREADS_DISTCLOCK 1000000
static __device__ int *d_DistClockArm;		// indexes to page number, from 0 to TOTAL_N_PAGES-1
static __device__ int *d_pageMapDistClock;

/* initialize TOTAL_N_PAGES pages on GPU's global memory, each page is PAGE_SIZE large
also initialize the page map structure with all 0 (all free)
randomize a number for each clock arm
 */
static __global__ void randomizeArmDistClock(){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid>=MAX_THREADS_DISTCLOCK) return;
	int16_t Clock = (int16_t)clock();
	int seed = (tid<<15) + Clock;
	seed = RNG_LCG(seed);
	seed = seed % TOTAL_N_PAGES;
	d_DistClockArm[tid] = seed;
}

__host__ void initPagesDistClock(){
	// initialize actual pages
	initPages();
	// initialize page map
	void *h_pageMapDistClock;
	gpuErrchk( hipMalloc((void**)&h_pageMapDistClock, TOTAL_N_PAGES*sizeof(int)) );
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_pageMapDistClock), &h_pageMapDistClock, sizeof(void*)) );
	gpuErrchk( hipMemset(h_pageMapDistClock, 0, TOTAL_N_PAGES*sizeof(int)) );
	// randomize arms
	void *_d_DistClockArm;
	gpuErrchk( hipMalloc((void**)&_d_DistClockArm, MAX_THREADS_DISTCLOCK*sizeof(int)) );
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_DistClockArm), &_d_DistClockArm, sizeof(void*)) );
	randomizeArmDistClock <<< ceil((float)MAX_THREADS_DISTCLOCK/32), 32 >>> ();
}

__device__ int getPageDistClock(int *step_count){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	bool foundFreePage = false;
	int pageID;
	int stepCount = 0;
	while (!foundFreePage){
		stepCount++;
		pageID = d_DistClockArm[tid]%TOTAL_N_PAGES;
		// check if page at current arm is free
		if (d_pageMapDistClock[pageID]==0){
			// try to mark page as taken
			int old_val = atomicExch(&(d_pageMapDistClock[pageID]), 1);
			if (old_val == 0)
				// this thread got the page
				foundFreePage = true;
		}
		// move arm
		d_DistClockArm[tid] = pageID + 1;
	}
	if (step_count) *step_count = stepCount;
	return pageID;
}

__device__ void freePageDistClock(int pageID, int *step_count){
	atomicExch(&(d_pageMapDistClock[pageID]), 0);
}

__global__ void printNumPagesLeftDistClock_kernel(){
	int count = 0;
	for (int i=0; i<TOTAL_N_PAGES; i++){
		if (d_pageMapDistClock[i]==0) count++;
	}
	printf("[DC info] Number of free pages: %d \n", count);
}
__host__ void printNumPagesLeftDistClock(){
	printNumPagesLeftDistClock_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

__global__ void resetBufferDistClock_kernel(){
	memset(d_pageMapDistClock, 0, TOTAL_N_PAGES*sizeof(int));
}
__host__ void resetBufferDistClock(){
	resetBufferDistClock_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}



/* ------------- COLLABORATIVE RANDOM WALK IMPLEMENTATION --------------------- */

/* Arrange the N elements of ARRAY in random order.
   Only effective if N is much smaller than RAND_MAX;
   if this may not be the case, use a better random
   number generator. */

/* function to replace hiprand, better efficiency */
#define WARPSIZE 32

__device__ int *d_PageMapCollabRW;      // length TOTAL_N_PAGES

/* initialize TOTAL_N_PAGES pages on GPU's global memory, each page is PAGE_SIZE large
also initialize the page map structure with all 0 (all free) */
__host__ void initPagesCollabRW(){
	// initialize actual pages
	initPages();

	// initialize page map of length TOTAL_N_PAGES
	void *h_PageMapCollabRW;
	gpuErrchk( hipMalloc((void**)&h_PageMapCollabRW, TOTAL_N_PAGES*sizeof(int)) );
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_PageMapCollabRW), &h_PageMapCollabRW, sizeof(void*)) );
	gpuErrchk( hipMemset(h_PageMapCollabRW, 0, TOTAL_N_PAGES*sizeof(int)) );
	
}

// if step_count is not null, write step count to it
__device__ int getPageCollabRW(int *stepCount){
	// initialize shared memory
	__shared__ int S_pageIDs_block[1024];  // this is to signal compiler to allocate enough for the entire block
	__shared__ int S_NPagesFound_blk[32];  
	int *S_pageIDs; 			// this is the shared array for the warp, calculated below
	int warpID = threadIdx.x>>5;
	S_pageIDs = &S_pageIDs_block[warpID<<5];
	int *S_NPagesFound = &S_NPagesFound_blk[warpID];    // keep track of number of pages found per warp
	if (threadIdx.x==0) *S_NPagesFound = 0;
	// find number of requests
	int NRequests = __popc(__activemask());

	// all active threads try find a random page every round
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int seed = (tid<<15) + ((int16_t)clock());
	int stepCount_ = 0;
	// variable meaning:
		// S_NPagesFound: warp-shared counter of number of pages found 
	while (*S_NPagesFound<NRequests) {
		stepCount_++;
		// find a random page
		seed = RNG_LCG(seed);
		unsigned pageID_ = (unsigned)(seed % TOTAL_N_PAGES);
		// check if page is available
		if (d_PageMapCollabRW[pageID_]==0){
			// if page is available, try to grab it
			int result = atomicExch(&d_PageMapCollabRW[pageID_],1);
			// if successful, add 1 to S_NPagesFound and add pageID to S_pageIDs
			if (result==0){
				int pos = atomicAdd(S_NPagesFound, 1);
				// if we have found more than what we need, return the page
				if (pos>=NRequests)
					freePageCollabRW(pageID_);
				// otherwise, add the pageID to the shared array
				else 
					S_pageIDs[pos] = pageID_;
			}
		}
	}

	// we have found enough pages and stored their IDs on shared memory
	// now grab them from shared memory
	int pos = atomicSub(S_NPagesFound, 1) - 1;
	int pageID = S_pageIDs[pos];

	if (stepCount) *stepCount = stepCount_;
	return pageID;
}


__device__ void freePageCollabRW(int pageID){
	atomicAnd(&d_PageMapCollabRW[pageID], 0);
}


__global__ void resetBufferCollabRW_kernel(){
	// set page map to 0
	memset(d_PageMapCollabRW, 0, TOTAL_N_PAGES*sizeof(int));
}

__host__ void resetBufferCollabRW(){
	resetBufferCollabRW_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}

__global__ void printNumPagesLeftCollabRW_kernel(){
	int count = 0;
	for (int i=0; i<TOTAL_N_PAGES; i++){
		if (d_PageMapCollabRW[i]==0) count++;
	}
	printf("[CoRW info] Number of free pages: %d \n", count);
}

__host__ void printNumPagesLeftCollabRW(){
	printNumPagesLeftCollabRW_kernel <<< 1, 1 >>> ();
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}
